#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

const int N = 1024;
const int BLOCK_SIZE = 32;

__global__ void blockedMatrixMultiplyKernel(const int *A, const int *B, int *C, int n) {
    __shared__ int sA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int sB[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * BLOCK_SIZE + ty;
    int col = bx * BLOCK_SIZE + tx;

    int sum = 0;

    for (int m = 0; m < n / BLOCK_SIZE; ++m) {
        sA[ty][tx] = A[row * n + (m * BLOCK_SIZE + tx)];
        sB[ty][tx] = B[(m * BLOCK_SIZE + ty) * n + col];
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += sA[ty][k] * sB[k][tx];
        }
        __syncthreads();
    }

    if (row < n && col < n) {
        C[row * n + col] = sum;
    }
}

void initializeMatrix(std::vector<int>& matrix) {
    for (int i = 0; i < N * N; i++) {
        matrix[i] = i % 10; // Simple initialization
    }
}

void cudaBlockedMatrixMultiply(const std::vector<int>& A, const std::vector<int>& B, std::vector<int>& C) {
    int *d_A, *d_B, *d_C;
    int size = N * N * sizeof(int);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), size, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N / BLOCK_SIZE, N / BLOCK_SIZE);

    blockedMatrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C.data(), d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void printPartialMatrix(const std::vector<int>& matrix) {
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            std::cout << matrix[i * N + j] << "\t";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
    std::vector<int> A(N * N), B(N * N), C(N * N, 0);

    initializeMatrix(A);
    initializeMatrix(B);

    std::cout << "Partial Matrix A:" << std::endl;
    printPartialMatrix(A);
    std::cout << "Partial Matrix B:" << std::endl;
    printPartialMatrix(B);

    auto start = std::chrono::high_resolution_clock::now();

    cudaBlockedMatrixMultiply(A, B, C);

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    std::cout << "Partial Result Matrix C:" << std::endl;
    printPartialMatrix(C);

    std::cout << "Total computation time: " << duration.count() << " milliseconds" << std::endl;

    return 0;
}
